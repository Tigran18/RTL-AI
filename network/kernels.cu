#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void apply_activation(double* input, double* output, int size, int act_type) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        switch (act_type) {
            case 0: // Sigmoid
                output[idx] = 1.0 / (1.0 + exp(-input[idx]));
                break;
            case 1: // ReLU
                output[idx] = max(0.0, input[idx]);
                break;
            case 2: // Tanh
                output[idx] = tanh(input[idx]);
                break;
        }
    }
}

__global__ void batch_norm(double* input, double* output, double mean, double variance,
                           double gamma, double beta, double epsilon, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = gamma * (input[idx] - mean) / sqrt(variance + epsilon) + beta;
    }
}

extern "C" void compute_activation_on_gpu(double* input, double* output, int size, int act_type) {
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    apply_activation<<<gridSize, blockSize>>>(input, output, size, act_type);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error in compute_activation_on_gpu at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
    }
}

extern "C" void compute_batch_norm_on_gpu(double* input, double* output, double mean, double variance, double gamma, double beta, double epsilon, int size) {
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    batch_norm<<<gridSize, blockSize>>>(input, output, mean, variance, gamma, beta, epsilon, size);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error in compute_batch_norm_on_gpu at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
    }
}